#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <iostream>

using namespace std;

__global__ void matrixMul(int *a, int *b, int N, int M)
{
    // Calculate the global row and column for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check for our matrix
    if(row < N && col < M)
    {
        // Accumulate a partial result
        int tmp = 0;
        for(int i = 0; i < N; i++)
        {
            //tmp += a[row * N + i] * b[i * N + col];
            tmp += a[i * M + col];
        }

        // Write back the result
        b[col] = tmp;
    }

    int max = 0;
    int index = 0;
    int max_index = 0;

    for (int k ; k<M;k++)
    {
        if (b[i]>max)
        {
            max = b[i];
            index = i;
        }
    }

    max_index = index;

}
// Initializes a square matrix with random numbers between 0-100
void init_matrix(int *m, int N, int M)
{
    for(int i = 0; i < N * M; i++)
    {
        m[i] = rand() % 100;

    }
}

// Verify the result on the CPU
void verify_result(int *a, int *b, int *c, int N){
    int tmp;
    // For every row...
    for(int i = 0; i < N; i++){
        // For every col...
        for(int j = 0; j < N; j++){
            // For every element in the row-col pair
            tmp = 0;
            for(int k = 0; k < N; k++){
                tmp += a[i * N + k] * b[k * N + j];
            }
            
            // Check each result
            assert(tmp == c[i * N + j]);
        }
    }
}

int main(){
    // Set our square matrix dimension (2^10 x 2^10 default) 
    int N = 1000;
    int M = 100;
    size_t bytes = N * M * sizeof(int);

    // Allocate memory for our matrices
    int *a, *b;
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    

    // Initialize our matrices
    init_matrix(a, N, M);
    init_matrix(b, 0, M);

    // Set our CTA and Grid dimensions
    int threads = 16;
    int blocks = (N + threads - 1) / threads;

    // Setup our kernel launch parameters
    dim3 THREADS(threads, threads);
    dim3 BLOCKS(blocks, blocks);

    // Launch our kernel
    matrixMul<<<BLOCKS, THREADS>>>(a, b, N, M);
    hipDeviceSynchronize();

    // Verify the result
    verify_result(a, b, N, M);

    cout << "PROGRAM COMPLETED SUCCESSFULLY!" << endl;
    
    // Free allocated memory
    hipFree(a);
    hipFree(b);
 
    return 0;
}
